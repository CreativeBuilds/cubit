#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include "uint256.cuh"
#include "main.hh"
#include <dirent.h>
#include <ctype.h>

// Copyright (c) 2009-2010 Satoshi Nakamoto
// Copyright (c) 2009-2019 The Bitcoin Core developers
// Distributed under the MIT software license, see the accompanying
// file COPYING or http://www.opensource.org/licenses/mit-license.php.
__device__ int lt(uint256 a, uint256 b) {
    // Assumes a and b are little-endian
    // This is correct for CUDA
    // https://stackoverflow.com/questions/15356622/anyone-know-whether-nvidias-gpus-are-big-or-little-endian
    
    for (int i = 8 - 1; i >= 0; i--) {
        if (a[i] < b[i]) {
            return -1;
        } else if (a[i] > b[i]) {
            return 1;
        }
    }
    return 0;
}

__device__ void sha256(unsigned char* data, unsigned long size, unsigned char* digest) {
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, data, size);
    sha256_final(&ctx, digest);
}

__device__ bool seal_meets_difficulty(BYTE* seal, uint256 limit) {
    // Need a 256 bit integer to store the seal number
    uint256 seal_number;
    
    // Seal is big-endian, and we want little-endian
    for (int i = 0; i < 8; i++) {
        seal_number[i] = seal[60 - (i * 4)] | (seal[61 - (i * 4)] << 8) | (seal[62 - (i * 4)] << 16) | (seal[63 - (i * 4)] << 24);
    }

    // Check if the seal number is less than the limit
    return lt(seal_number, limit) == -1;
}

__device__ void create_seal_hash(BYTE* seal, BYTE* block_hash, unsigned long nonce) {
    BYTE pre_seal[40];
    
    // Convert nonce to bytes (little endian) and store at start of pre_seal;
    for (int i = 0; i < 8; i++) {
        pre_seal[i] = (nonce >> (i * 8)) & 0xFF;
    }

    // Store the block bytes at the end of pre_seal;
    for (int i = 0; i < 32; i++) {
        pre_seal[i + 8] = block_hash[i];
    }
    
    // Hash the pre_seal and store in seal;
    sha256(pre_seal, sizeof(BYTE) * 40, seal);     
}

__global__ void solve(BYTE* seal, unsigned long* solution, unsigned long* nonce_start, unsigned long update_interval, unsigned int n_nonces, uint256 limit, BYTE* block_bytes) {
        BYTE seal_[64];
        
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
                i < n_nonces; 
                i += blockDim.x * gridDim.x) 
            {
                unsigned long nonce = nonce_start[i];
                for (unsigned long j = nonce; j < nonce + update_interval; j++) {
                    create_seal_hash(seal_, block_bytes, j);
                    if (seal_meets_difficulty(seal_, limit)) {
                        solution[i] = j;
                        // Copy seal to shared memory
                        for (int k = 0; k < 64; k++) {
                            seal[k] = seal_[k];
                        }
                        break;
                    }            
                }
            }            
}

__global__ void test_sha256(BYTE* data, int size, BYTE* digest) {
    sha256(data, size, digest);
}

__global__ void test_seal_hash(BYTE* seal, BYTE* block_hash, unsigned long nonce) {
    create_seal_hash(seal, block_hash, nonce);
}

__global__ void test_preseal_hash(BYTE* seal, BYTE* preseal_bytes) {
    sha256(preseal_bytes, sizeof(BYTE) * 40, seal);
}

void pre_sha256() {
	// copy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void runSolve(int blockSize, BYTE* seal, unsigned long* solution, unsigned long* nonce_start, unsigned long update_interval, unsigned int n_nonces, uint256 limit, BYTE* block_bytes) {
	int numBlocks = (n_nonces + blockSize - 1) / blockSize;
	solve <<< numBlocks, blockSize >>> (seal, solution, nonce_start, update_interval, n_nonces, limit, block_bytes);
}

void runTest(BYTE* data, unsigned long size, BYTE* digest) {
    // Test sha256
    BYTE* dev_data;
    BYTE* dev_digest;
    checkCudaErrors(hipMallocManaged(&dev_data, size));
    checkCudaErrors(hipMallocManaged(&dev_digest, sizeof(BYTE) * 64));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_data, data, size, hipMemcpyHostToDevice));

    pre_sha256();
    test_sha256<<<1, 1>>>(dev_data, size, dev_digest);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(digest, dev_digest, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTestSealHash(BYTE* seal, BYTE* block_hash, unsigned long nonce) {
    BYTE* dev_seal;
    BYTE* dev_block_hash;
    checkCudaErrors(hipMallocManaged(&dev_seal, 64));
    checkCudaErrors(hipMallocManaged(&dev_block_hash, 32));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_block_hash, block_hash, 32, hipMemcpyHostToDevice));

    pre_sha256();

    test_seal_hash<<<1, 1>>>(dev_seal, dev_block_hash, nonce);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(seal, dev_seal, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTestPreSealHash(unsigned char* seal, unsigned char* preseal_bytes) {
    BYTE* dev_seal;
    BYTE* dev_preseal_bytes;
    checkCudaErrors(hipMallocManaged(&dev_seal, 64));
    checkCudaErrors(hipMallocManaged(&dev_preseal_bytes, 40));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_preseal_bytes, preseal_bytes, 40, hipMemcpyHostToDevice));

    pre_sha256();

    test_preseal_hash<<<1, 1>>>(dev_seal, dev_preseal_bytes);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(seal, dev_seal, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

unsigned long solve_cuda_c(int blockSize, BYTE* seal, unsigned long* nonce_start, unsigned long update_interval, unsigned int n_nonces, uint256 limit, BYTE* block_bytes) {
	unsigned long* nonce_start_d;
	unsigned char* block_bytes_d;
    BYTE* seal_d;
    unsigned long* solution_d;
    unsigned long solution[1] = {0};
    unsigned long* limit_d;

    // Allocate memory on device
    
    // Malloc space for solution in device memory. Should be a single unsigned long.
    printf("Allocating memory on device\n");
    checkCudaErrors(hipMallocManaged(&solution_d, sizeof(unsigned long)));
    // Malloc space for seal in device memory. Should be one seal.
    checkCudaErrors(hipMallocManaged(&seal_d, 64 * sizeof(BYTE)));
    // Malloc space for nonce_start in device memory.
    checkCudaErrors(hipMallocManaged(&nonce_start_d, n_nonces * sizeof(unsigned long)));
    // Malloc space for block_bytes in device memory. Should be 32 bytes.
    checkCudaErrors(hipMallocManaged(&block_bytes_d, 32 * sizeof(BYTE)));
    // Malloc space for limit in device memory.
    checkCudaErrors(hipMallocManaged(&limit_d, 8 * sizeof(unsigned long)));

	// Copy data to device memory
    printf("Copying memory to device\n");
	// Put block bytes in device memory. Should be 32 bytes.
	checkCudaErrors(hipMemcpy(block_bytes_d, block_bytes, 32 * sizeof(BYTE), hipMemcpyHostToDevice));
	// Put nonce_start in device memory. Should be a single int for each thread.
	checkCudaErrors(hipMemcpy(nonce_start_d, nonce_start, n_nonces * sizeof(unsigned long), hipMemcpyHostToDevice));
    // Put limit in device memory.
    checkCudaErrors(hipMemcpy(limit_d, limit, 8 * sizeof(unsigned long), hipMemcpyHostToDevice));

    // Set seal to 0xff
    checkCudaErrors(hipMemset(seal_d, 0xff, 64 * sizeof(unsigned char)));
    // Zero solution
    solution_d[0] = 0;

	pre_sha256();

    // Running Solve on GPU
    printf("Running solve on GPU\n");
	runSolve(blockSize, seal_d, solution_d, nonce_start_d, update_interval, n_nonces, limit_d, block_bytes_d);

	hipDeviceSynchronize();
    
    // Copy data back to host memory
    printf("Copying memory to host\n");
    checkCudaErrors(hipMemcpy(solution, solution_d, 1 * sizeof(unsigned long), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(seal, seal_d, 64 * sizeof(BYTE), hipMemcpyDeviceToHost));
    
	hipDeviceReset();
	return solution[0];
}

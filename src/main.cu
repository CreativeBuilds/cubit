#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include "uint256.cuh"
#include "uint64.cuh"
#include "main.hh"
#include <dirent.h>
#include <ctype.h>

// Copyright (c) 2009-2010 Satoshi Nakamoto
// Copyright (c) 2009-2019 The Bitcoin Core developers
// Distributed under the MIT software license, see the accompanying
// file COPYING or http://www.opensource.org/licenses/mit-license.php.
__device__ int lt(uint256 a, uint256 b) {
    // Check if a is less than b
    // Assumes a and b are little-endian
    // This is correct for CUDA
    // https://stackoverflow.com/questions/15356622/anyone-know-whether-nvidias-gpus-are-big-or-little-endian
    BYTE* a_ = (BYTE*)a;
    BYTE* b_ = (BYTE*)b;

    for (int i = 32 - 1; i >= 0; i--) {
        if (a_[i] < b_[i]) {
            return -1;
        } else if (a_[i] > b_[i]) {
            return 1;
        }
    }
    return 0;
}

__device__ void sha256(unsigned char* data, unsigned long size, unsigned char* digest) {
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, data, size);
    sha256_final(&ctx, digest);
}

__device__ bool seal_meets_difficulty(BYTE* seal, uint256 limit) {
    // Need a 256 bit integer to store the seal number
    BYTE seal_[32];

    // Reverse 32 byte array to get little-endian
    for (int i = 0; i < 32; i++) {
        seal_[i] = seal[31-i];
    }

    // Check if the seal number is less than the limit
    int under_limit = lt((unsigned long*)seal_, limit);
    return under_limit == -1;
}

__device__ void create_nonce_bytes(uint64 nonce, BYTE* nonce_bytes) {
    // Convert nonce to bytes (little endian) and store at start of pre_seal;
    for (int i = 0; i < 4; i++) {
        nonce_bytes[i] = (nonce >> (i * 8)) & 0xFF;
    }
}

__device__ int convert_from_ascii_to_int(BYTE ascii_bytes) {
    // Convert the ascii bytes to an integer
    // The ascii bytes are in the form of a hexadecimal number
    int result = 0;
    if (ascii_bytes >= '0' && ascii_bytes <= '9') {
        result = ascii_bytes - '0';
    } else if (ascii_bytes >= 'a' && ascii_bytes <= 'f') {
        result = ascii_bytes - 'a' + 10;
    }
    return result;
}

__device__ void create_pre_seal(BYTE* pre_seal, BYTE* block_hash_bytes, uint64 nonce) {
    BYTE pre_pre_seal[40];
    create_nonce_bytes(nonce, pre_pre_seal);

    for (int i = 0; i < 32; i += 1) {
        // Convert each into ascii and then hex
        unsigned char high_bits = block_hash_bytes[2*i];
        unsigned char low_bits = block_hash_bytes[2*i+1];
        pre_pre_seal[i + 8] = convert_from_ascii_to_int(high_bits) * 16 + convert_from_ascii_to_int(low_bits);
    }

    for (int i = 0; i < 40; i++) {
        pre_seal[i] = pre_pre_seal[i];
    }
}

__device__ void create_seal_hash(BYTE* seal, BYTE* block_hash, uint64 nonce) {
    BYTE pre_seal[40];    

    create_pre_seal(pre_seal, block_hash, nonce);
    
    // Hash the pre_seal and store in seal;
    sha256(pre_seal, sizeof(BYTE) * 40, seal);     
}

__global__ void solve(BYTE** seals, uint64* solution, uint64 nonce_start, uint64 update_interval, unsigned int n_nonces, uint256 limit, BYTE* block_bytes) {
        __shared__ bool found;
        found = false;
        
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; 
                i < n_nonces; 
                i += blockDim.x * gridDim.x) 
            {
                if (found) {
                    break;
                }
                BYTE seal[64];

                // Make the seal all 0xff
                for (int j = 0; j < 64; j++) {
                    seal[i] = 0xff;
                }

                uint64 nonce = nonce_start + i * update_interval;
                for (
                    uint64 j = nonce; j < nonce + update_interval; j++) {
                    create_seal_hash(seal, block_bytes, j);
                    
                    if (seal_meets_difficulty(seal, limit)) {
                        solution[i] = j + 1;

                        // Copy seal to shared memory
                       // for (int k = 0; k < 64; k++) {
                         //   seal[i][k] = seal_[k];
                            // print the seal
                            //if (k == 32) {
                            //    printf("i = 32;\n");
                            //}
                            //printf("%02x ", seal_[k]);
                        
                        //}
                        //printf("\n");
                        found = true;
                        break;
                    }
                }
            }            
}

__global__ void test_lt(uint256 a, uint256 b, int* result) {
    result[0] = lt(a, b);
}

__global__ void test_create_nonce_bytes(uint64 nonce, BYTE* nonce_bytes) {
    create_nonce_bytes(nonce, nonce_bytes);
}

__global__ void test_create_preseal(BYTE* pre_seal, uint64 nonce, BYTE* block_bytes) {
    create_pre_seal(pre_seal, block_bytes, nonce);
}

__global__ void test_sha256(BYTE* data, int size, BYTE* digest) {
    sha256(data, size, digest);
}

__global__ void test_seal_hash(BYTE* seal, BYTE* block_hash, uint64 nonce) {
    create_seal_hash(seal, block_hash, nonce);
}

__global__ void test_preseal_hash(BYTE* seal, BYTE* preseal_bytes) {
    sha256(preseal_bytes, sizeof(BYTE) * 40, seal);
}

__global__ void test_seal_meets_difficulty(BYTE* seal, uint256 limit, bool* result) {
    seal_meets_difficulty(seal, limit);
}

void pre_sha256() {
	// copy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

void runSolve(int blockSize, BYTE** seals, uint64* solution, uint64 nonce_start, uint64 update_interval, uint256 limit, BYTE* block_bytes) {
	int numBlocks = (blockSize + blockSize - 1) / blockSize;
	solve <<< numBlocks, blockSize >>> (seals, solution, nonce_start, update_interval, blockSize, limit, block_bytes);
}

bool runTestSealMeetsDifficulty(BYTE* seal, uint256 limit) {
    BYTE* dev_seal;
    unsigned long* dev_limit;
    bool* dev_result;

    bool result;

    checkCudaErrors(hipMallocManaged(&dev_seal, sizeof(BYTE) * 32));
    checkCudaErrors(hipMallocManaged(&dev_limit, 8 * sizeof(unsigned long)));
    checkCudaErrors(hipMallocManaged(&dev_result, sizeof(bool)));

    checkCudaErrors(hipMemcpy(dev_seal, seal, sizeof(BYTE) * 32, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_limit, limit, 8 * sizeof(unsigned long), hipMemcpyHostToDevice));
    
    test_seal_meets_difficulty <<< 1, 1 >>> (seal, limit, dev_result);

    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(&result, dev_result, sizeof(bool), hipMemcpyDeviceToHost));
    hipDeviceReset();

    return result;
}

void runTestCreateNonceBytes(uint64 nonce, BYTE* nonce_bytes) {
    BYTE* dev_nonce_bytes;
    checkCudaErrors(hipMallocManaged(&dev_nonce_bytes, sizeof(BYTE) * 8));

    pre_sha256();

    test_create_nonce_bytes<<<1, 1>>>(nonce, dev_nonce_bytes);

    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(nonce_bytes, dev_nonce_bytes, sizeof(BYTE) * 8, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTestCreatePreSeal(unsigned char* pre_seal, uint64 nonce, unsigned char* block_bytes) {
    // Test sha256
    BYTE* dev_pre_seal;
    checkCudaErrors(hipMallocManaged(&dev_pre_seal, sizeof(BYTE) * 40));

    // malloc block_bytes
    BYTE* dev_block_bytes;
    checkCudaErrors(hipMallocManaged(&dev_block_bytes, 64 * sizeof(BYTE)));
    checkCudaErrors(hipMemcpy(dev_block_bytes, block_bytes, 64 *  sizeof(BYTE), hipMemcpyHostToDevice));

    pre_sha256();

    test_create_preseal<<<1, 1>>>(dev_pre_seal, nonce, dev_block_bytes);

    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(pre_seal, dev_pre_seal, sizeof(BYTE) * 40, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTest(BYTE* data, unsigned long size, BYTE* digest) {
    // Test sha256
    BYTE* dev_data;
    BYTE* dev_digest;
    checkCudaErrors(hipMallocManaged(&dev_data, size));
    checkCudaErrors(hipMallocManaged(&dev_digest, sizeof(BYTE) * 64));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_data, data, size, hipMemcpyHostToDevice));

    pre_sha256();
    test_sha256<<<1, 1>>>(dev_data, size, dev_digest);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(digest, dev_digest, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTestSealHash(BYTE* seal, BYTE* block_hash, uint64 nonce) {
    BYTE* dev_seal;
    BYTE* dev_block_hash;
    checkCudaErrors(hipMallocManaged(&dev_seal, 64));
    checkCudaErrors(hipMallocManaged(&dev_block_hash, 64));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_block_hash, block_hash, 64, hipMemcpyHostToDevice));

    pre_sha256();

    test_seal_hash<<<1, 1>>>(dev_seal, dev_block_hash, nonce);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(seal, dev_seal, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

void runTestPreSealHash(unsigned char* seal, unsigned char* preseal_bytes) {
    BYTE* dev_seal;
    BYTE* dev_preseal_bytes;
    checkCudaErrors(hipMallocManaged(&dev_seal, 64));
    checkCudaErrors(hipMallocManaged(&dev_preseal_bytes, 40));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_preseal_bytes, preseal_bytes, 40, hipMemcpyHostToDevice));

    pre_sha256();

    test_preseal_hash<<<1, 1>>>(dev_seal, dev_preseal_bytes);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(seal, dev_seal, 64, hipMemcpyDeviceToHost));
    hipDeviceReset();
}

int runTestLessThan(uint256 a, uint256 b) {
    unsigned long* dev_a;
    unsigned long* dev_b;
    int* dev_result;
    int result[1];
    checkCudaErrors(hipMallocManaged(&dev_a, 8 * sizeof(unsigned long)));
    checkCudaErrors(hipMallocManaged(&dev_b, 8 * sizeof(unsigned long)));
    checkCudaErrors(hipMallocManaged(&dev_result, 1 * sizeof(int)));
    // Copy data to device
    checkCudaErrors(hipMemcpy(dev_a, a, 8 * sizeof(unsigned long), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, 8 * sizeof(unsigned long), hipMemcpyHostToDevice));
    
    test_lt<<<1, 1>>>(dev_a, dev_b, dev_result);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(result, dev_result, sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceReset();

    return result[0];
}

uint64 solve_cuda_c(int blockSize, BYTE* seal, uint64 nonce_start, uint64 update_interval, uint256 limit, BYTE* block_bytes) {
	unsigned char* block_bytes_d;
    unsigned char* block_bytes_h;
    uint64* solution_d;
    uint64* solutions;
    uint64 solution = 0;
    unsigned long* limit_d;
    unsigned long* limit_h;

    // Allocate pinned memory on host. This should speed up the data transfer back.
    checkCudaErrors(hipHostMalloc((void**)&solutions, blockSize * 8 * sizeof(unsigned long)));
    checkCudaErrors(hipHostMalloc((void**)&block_bytes_h, 64 * sizeof(BYTE)));
    checkCudaErrors(hipHostMalloc((void**)&limit_h, 8 * sizeof(unsigned long)));
    // Copy into pinned memory
    memcpy(block_bytes_h, block_bytes, 64 * sizeof(BYTE));
    memcpy(limit_h, limit, 8 * sizeof(unsigned long));
    // Allocate memory on device
    
    // Malloc space for solution in device memory. Should be a single unsigned long.
    checkCudaErrors(hipMallocManaged(&solution_d, blockSize * sizeof(uint64)));
    // Malloc space for block_bytes in device memory. Should be 32 bytes.
    checkCudaErrors(hipMallocManaged(&block_bytes_d, 64 * sizeof(BYTE)));
    // Malloc space for limit in device memory.
    checkCudaErrors(hipMallocManaged(&limit_d, 8 * sizeof(unsigned long)));

	// Copy data to device memory
	// Put block bytes in device memory. Should be 32 bytes.
	checkCudaErrors(hipMemcpy(block_bytes_d, block_bytes_h, 64 * sizeof(BYTE), hipMemcpyHostToDevice));
    // Put limit in device memory.
    checkCudaErrors(hipMemcpy(limit_d, limit_h, 8 * sizeof(unsigned long), hipMemcpyHostToDevice));

	pre_sha256();

    // Zero out solution
    checkCudaErrors(hipMemset(solution_d, 0, sizeof(uint64)));

    // Running Solve on GPU
	runSolve(blockSize, NULL, solution_d, nonce_start, update_interval, limit_d, block_bytes_d);

	hipDeviceSynchronize();
    
    // Copy data back to host memory
    checkCudaErrors(hipMemcpy(solutions, solution_d, blockSize * sizeof(uint64), hipMemcpyDeviceToHost));
    // Check if solution is valid
    for (int i = 0; i < blockSize; i++) {
        if (solutions[i] != 0) {
            solution = solutions[i];
            break;
        }
    }

    
    // Free memory
    checkCudaErrors(hipFree(solution_d));
    checkCudaErrors(hipFree(block_bytes_d));
    checkCudaErrors(hipFree(limit_d));

    checkCudaErrors(hipHostFree(solutions));
    checkCudaErrors(hipHostFree(block_bytes_h));
    checkCudaErrors(hipHostFree(limit_h));	
    return solution;
}


void reset_cuda_c() {
    hipDeviceReset();
}